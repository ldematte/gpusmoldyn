#include "hip/hip_runtime.h"
/*
 * GPU Smoldyn: Smoldyn algorithm ported to the GPU using CUDA 2.2
 * Writtern By Lorenzo Dematt�, 2010-2011
 *
 * This file is part of GPU Smoldyn
 * 
 *     GPU Smoldyn is free software: you can redistribute it and/or modify
 *     it under the terms of the GNU General Public License as published by
 *     the Free Software Foundation, either version 3 of the License, or
 *     (at your option) any later version.
 * 
 *     GPU Smoldyn is distributed in the hope that it will be useful,
 *     but WITHOUT ANY WARRANTY; without even the implied warranty of
 *     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *     GNU General Public License for more details.
 * 
 *     You should have received a copy of the GNU General Public License
 *     along with Foobar.  If not, see <http://www.gnu.org/licenses/>. 
 * 
 * Based on algorithm and source code of Smoldyn, written by Steve Andrews, 2003.
 * 
 * Portions taken by code examples in NVIDIA Whitepapers, GPU Gems 2 and 3, 
 * Copyright 1993-2009 NVIDIA Corporation, Addison-Wesley and the original authors. 
 * 
 */
 
#ifndef _PARTICLES_KERNEL_H_
#define _PARTICLES_KERNEL_H_

#include "cutil_math.h"
#include "hip/hip_math_constants.h"
#include "particles_kernel.cuh"
#include "surfaces_kernel.cuh"

#include "reaction.h"
#include "reaction_kernel.cuh"
#include "rand_kernel.cuh"

#include "MersenneTwister.h"


#if USE_TEX
// textures for particle position and velocity
texture<float4, 1, hipReadModeElementType> oldPosTex;
texture<float4, 1, hipReadModeElementType> oldVelTex;

texture<uint, 1, hipReadModeElementType> gridParticleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<uint, 1, hipReadModeElementType> cellEndTex;
#endif

texture<float, 1, hipReadModeElementType> diffusionRatesTex; //difstep

// simulation parameters in constant memory
__constant__ SimParams params;

__constant__ float gaussianLookupTable[gaussianTableDim];
const int gaussianTableDimMinusOne = gaussianTableDim -1;

__global__
void adjustPositions(float4* posArray,       // lenght = numParticles
                     int*    types,          // lenght = numParticles
                     uint    numParticles,
                     float   particleRadius)
{
    uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;          // handle case when no. of particles not multiple of block size

  	 float4 pos = posArray[index];    // ensure coalesced read
    //volatile int typeId = typeArray[index];
    
    // check collisions with impenetrable surfaces
    if (pos.x > 1.0f - particleRadius)  
       pos.x = 1.0f - particleRadius;
    if (pos.x < -1.0f + particleRadius)
       pos.x = -1.0f + particleRadius; 

    if (pos.y > 1.0f - particleRadius)
       pos.y = 1.0f - particleRadius; 
    if (pos.y < -1.0f + particleRadius) 
       pos.y = -1.0f + particleRadius; 

    if (pos.z > 1.0f - particleRadius)
       pos.z = 1.0f - particleRadius; 
    if (pos.z < -1.0f + particleRadius)
       pos.z = -1.0f + particleRadius; 

    

    // TODO: periodic boundaries

    // TODO: compute probability of absorbtion

    posArray[index] = pos;
}

__global__
void compactPosAndTypeD(int* dCompactedTemp, // [IN] array of compacted indexes (prefix scan)
                      int* deathBirth,     // [IN] array of live/death flags
                      int* dTypes,         // [IN/OUT] type array (of int) to compact
                      float* dPos,         // [IN/OUT] pos array (of float4) to compact
                      int numParticles)  // dimension of the 3 arrays
{
    uint i = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (i >= numParticles) return;          // handle case when no. of particles not multiple of block size
   
   //TODO: coalesced read/write?
   if (deathBirth[i] != 0) 
   {
      int newIndex = dCompactedTemp[i];
      dTypes[newIndex] = dTypes[i];

      // multiply by 4
      newIndex = newIndex * 4;
      i = i * 4;

      dPos[newIndex] = dPos[i];
      dPos[newIndex + 1] = dPos[i + 1];
      dPos[newIndex + 2] = dPos[i + 2];
      dPos[newIndex + 3] = dPos[i + 3];      
   }
   __syncthreads();
}

// integrate particle attributes
__global__
void diffuse(float4* posArray,  // input/output
               int* typeArray, // input/output
               //float4* velArray,  
               MersenneTwisterState* rngStateArray,
               float deltaTime,
               uint numParticles)
{
    uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;          // handle case when no. of particles not multiple of block size

	 volatile float4 posData = posArray[index];    // ensure coalesced read
    volatile int typeId = typeArray[index];

    int rngIndex = index % MT_RNG_COUNT;
    MersenneTwisterState* rngState = &(rngStateArray[rngIndex]);
    //volatile float4 velData = velArray[index];
    float3 pos = make_float3(posData.x, posData.y, posData.z);
    //float3 vel = make_float3(velData.x, velData.y, velData.z);

    //vel += params.gravity * deltaTime;
    //vel *= params.globalDamping;

    // TODO: compute new position !!
    //pos += vel * deltaTime;

    // requires 3 random numbers
    // difstep: pre-computed texture
    // gtable: gaussian table
    float rate = tex1Dfetch(diffusionRatesTex, typeId);
    int randX = MersenneTwisterGenerate(rngState, index) & gaussianTableDimMinusOne;
    int randY = MersenneTwisterGenerate(rngState, index) & gaussianTableDimMinusOne;
    int randZ = MersenneTwisterGenerate(rngState, index) & gaussianTableDimMinusOne;

    pos.x += rate * gaussianLookupTable[randX]; 
    pos.y += rate * gaussianLookupTable[randY]; 
    pos.z += rate * gaussianLookupTable[randZ]; 

    // store new position
    posArray[index] = make_float4(pos, posData.w);
}


// calculate position in uniform grid
__device__ int3 calcGridPos(float3 p)
{
    int3 gridPos;
    gridPos.x = floor((p.x - params.worldOrigin.x) / params.cellSize.x);
    gridPos.y = floor((p.y - params.worldOrigin.y) / params.cellSize.y);
    gridPos.z = floor((p.z - params.worldOrigin.z) / params.cellSize.z);
    return gridPos;
}

// calculate address in grid from position (clamping to edges)
__device__ uint calcGridHash(int3 gridPos)
{
    gridPos.x = gridPos.x & (params.gridSize.x-1);  // wrap grid, assumes size is power of 2
    gridPos.y = gridPos.y & (params.gridSize.y-1);
    gridPos.z = gridPos.z & (params.gridSize.z-1);        
    return __umul24(__umul24(gridPos.z, params.gridSize.y), params.gridSize.x) + __umul24(gridPos.y, params.gridSize.x) + gridPos.x;
}

// calculate grid hash value for each particle
__global__
void calcHashD(uint*   gridParticleHash,  // output
               uint*   gridParticleIndex, // output
               float4* pos,               // input: positions
               uint    numParticles)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;
    
    volatile float4 p = pos[index];

    // get address in grid
    int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
    uint hash = calcGridHash(gridPos);

    // store grid hash and particle index
    gridParticleHash[index] = hash;
    gridParticleIndex[index] = index;
}

// rearrange particle data into sorted order, and find the start of each cell
// in the sorted hash array
__global__
void reorderDataAndFindCellStartD(
                           uint*   cellStart,        // output: cell start index
							      uint*   cellEnd,          // output: cell end index
							      float4* sortedPos,        // output: sorted positions
  							      //float4* sortedVel,        // output: sorted velocities
                           uint *  gridParticleHash, // input: sorted grid hashes
                           uint *  gridParticleIndex,// input: sorted particle indices
				               float4* oldPos,           // input: sorted position array
							      //float4* oldVel,           // input: sorted velocity array
							      uint    numParticles)
{
	extern __shared__ uint sharedHash[];    // blockSize + 1 elements
    uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	
    uint hash;
    // handle case when no. of particles not multiple of block size
    if (index < numParticles) {
        hash = gridParticleHash[index];

        // Load hash data into shared memory so that we can look 
        // at neighboring particle's hash value without loading
        // two hash values per thread
	    sharedHash[threadIdx.x+1] = hash;

	    if (index > 0 && threadIdx.x == 0)
	    {
		    // first thread in block must load neighbor particle hash
		    sharedHash[0] = gridParticleHash[index-1];
	    }
	}

	__syncthreads();
	
	if (index < numParticles) {
		// If this particle has a different cell index to the previous
		// particle then it must be the first particle in the cell,
		// so store the index of this particle in the cell.
		// As it isn't the first particle, it must also be the cell end of
		// the previous particle's cell

	    if (index == 0 || hash != sharedHash[threadIdx.x])
	    {
		    cellStart[hash] = index;
            if (index > 0)
                cellEnd[sharedHash[threadIdx.x]] = index;
	    }

        if (index == numParticles - 1)
        {
            cellEnd[hash] = index + 1;
        }

	    // Now use the sorted index to reorder the pos and vel data
	    uint sortedIndex = gridParticleIndex[index];
	    float4 pos = FETCH(oldPos, sortedIndex);       // macro does either global read or texture fetch
       // float4 vel = FETCH(oldVel, sortedIndex);       // see particles_kernel.cuh

       sortedPos[index] = pos;
       // sortedVel[index] = vel;
   }
}

// collide two spheres using DEM method
__device__
float3 collideSpheres(float3 posA, float3 posB,
                      float3 velA, float3 velB,
                      float radiusA, float radiusB,
                      float attraction)
{
	// calculate relative position
    float3 relPos = posB - posA;

    float dist = length(relPos);
    float collideDist = radiusA + radiusB;

    float3 force = make_float3(0.0f);
    if (dist < collideDist) {
        float3 norm = relPos / dist;

		// relative velocity
        float3 relVel = velB - velA;

        // relative tangential velocity
        float3 tanVel = relVel - (dot(relVel, norm) * norm);

        // spring force
        force = -params.spring*(collideDist - dist) * norm;
        // dashpot (damping) force
        force += params.damping*relVel;
        // tangential shear force
        force += params.shear*tanVel;
		// attraction
        force += attraction*relPos;
    }

    return force;
}

inline __device__ float lengthSquared(float3 v)
{
    return dot(v, v);
}


// collide a particle against all other particles in a given cell
__device__
void collideCell(int3    gridPos,
                 uint    index,
                 float3  pos1,
                 float4* oldPos,
                 float4* newPos,
                 uint* gridParticleIndex,      //length = numParticles
                 int* types,
                 int numTypes,
                 int* birthDeath,
                 float* diffusionCoefficients,
                 int* reactionTable,
                 Reaction* reactionList,
                 uint*   cellStart,
                 uint*   cellEnd,
                 MersenneTwisterState* randState,
                 unsigned int threadId)
{
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = FETCH(cellStart, gridHash);

    uint mol1Index = gridParticleIndex[index];

    //if (birthDeath[mol1Index] >= 0) //this molecule was already processed
    //   return;
    //birthDeath[mol1Index] = 1; // we keep it, as a default

    int mol1Type = types[mol1Index];
    //invalid molecule?
    if (mol1Type <= 0)
    {
       birthDeath[mol1Index] = 0;// should never happen!
       return;
    }

    float3 force = make_float3(0.0f);
    if (startIndex != 0xffffffff) {        // cell is not empty
        // iterate over particles in this cell
        uint endIndex = FETCH(cellEnd, gridHash);
        for(uint j=startIndex; j<endIndex; j++) 
        {
           // index is the current particle index, j the one we will test on
           // both are indexes to the sorted array 

           // check not colliding with self
            if (j != index) 
            {
               // get back the original unsorted location
               uint mol2Index = gridParticleIndex[j];

               //if (birthDeath[mol2Index] >= 0) //this molecule was already processed
               //   continue;
               //birthDeath[mol2Index] = 1; //for now, flag as "save it"

               int mol2Type = types[mol2Index];
               if (mol2Type == -1)
               {
                  birthDeath[mol2Index] = 0;// should never happen!
                  continue;
               }

               // is a reaction possible between them?
               // TODO: multiple reactions possible!
               int tableIdx = mol1Type * numTypes + mol2Type;
               int r = reactionTable[tableIdx];
               if (r == -1)
                  continue;               
               
	            float3 pos2 = make_float3(FETCH(oldPos, j));

               // calculate relative position
               float3 relPos = pos1 - pos2;
               float distSquared = lengthSquared(relPos);
               
               if (distSquared <= reactionList->bindRadiusSquared[r])
               {
                  float p = reactionList->prob[r];
                  if (p == 1 || randReal(randState, threadId) <p)    
                  {
                     bimReact(reactionList, r, 
                        mol1Index, mol2Index,  
                        mol1Type, mol2Type, 
                        pos1, pos2, 
                        newPos, types, birthDeath,
                        gridParticleIndex,
                        diffusionCoefficients, 
                        randState, threadId); 
                  }
               }
            }
        }
    }
}

__global__ 
void checkSurfaceCollisions(float4* newPos,               // input: new positions
                            float4* oldPos,               // input: old positions
                            int* birthDeath,              // output: who was absorbed (dead) TODO check is handled by later steps
                            Panel* panels,
                            uint numParticles)
{
   uint molIndex = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
   if (molIndex >= numParticles) return;          // handle case when no. of particles not multiple of block size

   float4 molPos = newPos[molIndex];
   float4 molOldPos = oldPos[molIndex];
   
   uint2 gridIndex = getSurfaceGridIndex(molPos);
   uint startIndex = gridIndex.x, endIndex = gridIndex.y;
   // iterate over panels in this cell
   for(uint j = startIndex; j < endIndex; j++) 
   {
      Panel* panel = &(panels[j]);
      uint numCross = 2;
      float4 crossPoint = lineXpanel(pos, oldPos, panel, &numCross);     
      if(numCross < 2) 
      {
         //a panel was crossed, deal with it
         doSurfaceInteraction(molIndex, panels, j, crossPoint);
      }
   }
}


__global__
void collideD(float4* newPos,               // output: new positions
              float4* oldPos,               // input: sorted positions
              uint*   gridParticleIndex,    // input: sorted particle indices
              int* types,
              int numTypes,
              int* birthDeath,
              float* diffusionCoefficients,
              int* reactionTable,
              Reaction* reactionList,
              uint*   cellStart,
              uint*   cellEnd,
              uint    numParticles, 
              MersenneTwisterState* rngStateArray)
{
    uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numParticles)
       return;

    int rngIndex = index % MT_RNG_COUNT;
    MersenneTwisterState* rngState = &(rngStateArray[rngIndex]);
    
    // read particle data from sorted arrays
	 float3 pos = make_float3(FETCH(oldPos, index));

    // get address in grid
    int3 gridPos = calcGridPos(pos);

    // examine neighbouring cells
    float3 force = make_float3(0.0f);
    for(int z=-1; z<=1; z++) 
    {
        for(int y=-1; y<=1; y++) 
        {
            for(int x=-1; x<=1; x++) 
            {
                int3 neighbourPos = gridPos + make_int3(x, y, z);
                collideCell(neighbourPos, index, pos,
                            oldPos, newPos, gridParticleIndex, 
                            types, numTypes, birthDeath, 
                            diffusionCoefficients, 
                            reactionTable, reactionList,
                            cellStart, cellEnd,
                            rngState, index);
            }
        }
    }

    // collide with cursor sphere
    //force += collideSpheres(pos, params.colliderPos, vel, make_float3(0.0f, 0.0f, 0.0f), params.particleRadius, params.colliderRadius, 0.0f);

    // write new velocity back to original unsorted location
    //uint originalIndex = gridParticleIndex[index];
    //newVel[originalIndex] = make_float4(vel + force, 0.0f);
}

__global__
void resetNonInteracting(int* birthDeath, int numParticles)
{
   uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
   if (index >= numParticles)
      return;

   if (birthDeath[index] == -1)
      birthDeath[index] = 1;
}


#endif
